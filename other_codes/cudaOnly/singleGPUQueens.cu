#include <hip/hip_runtime.h>
#include <stdio.h>

#define _QUEENS_BLOCK_SIZE_ 	128
#define _EMPTY_      -1

typedef struct queen_root{
    unsigned int control;
    int8_t board[12];
} QueenRoot;


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


inline void prefixesHandleSol(QueenRoot *root_prefixes,unsigned int flag,char *board,int initialDepth,int num_sol){

    root_prefixes[num_sol].control = flag;

    for(int i = 0; i<initialDepth;++i)
      root_prefixes[num_sol].board[i] = (char)board[i];
}

inline bool MCstillLegal(const char *board, const int r)
{
    
    int i;
    int ld;
    int rd;
  // Check vertical
  for ( i = 0; i < r; ++i)
    if (board[i] == board[r]) return false;
    // Check diagonals
    ld = board[r];  //left diagonal columns
    rd = board[r];  // right diagonal columns
    for ( i = r-1; i >= 0; --i) {
      --ld; ++rd;
      if (board[i] == ld || board[i] == rd) return false;
    }

    return true;
}

__device__  bool GPU_queens_stillLegal(const char *board, const int r){

  bool safe = true;
  int i;
  register int ld;
  register int rd;
  // Check vertical
  for ( i = 0; i < r; ++i)
    if (board[i] == board[r]) safe = false;
    // Check diagonals
    ld = board[r];  //left diagonal columns
    rd = board[r];  // right diagonal columns
    for ( i = r-1; i >= 0; --i) {
      --ld; ++rd;
      if (board[i] == ld || board[i] == rd) safe = false;
    }

    return safe;
}


__global__ void BP_queens_root_dfs(int N, unsigned int nPreFixos, int depthPreFixos,
    QueenRoot *root_prefixes,unsigned long long int *vector_of_tree_size, unsigned long long int *sols){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nPreFixos) {
        register unsigned int flag = 0;
        register unsigned int bit_test = 0;
        register char vertice[20]; //representa o ciclo
        register int N_l = N;
        register int i, depth; 
        register unsigned long long  qtd_solucoes_thread = 0ULL;
        register int depthGlobal = depthPreFixos;
        register unsigned long long int tree_size = 0ULL;

        #pragma unroll 2
        for (i = 0; i < N_l; ++i) {
            vertice[i] = _EMPTY_;
        }

        flag = root_prefixes[idx].control;

        #pragma unroll 2
        for (i = 0; i < depthGlobal; ++i)
            vertice[i] = root_prefixes[idx].board[i];

        depth=depthGlobal;

        do{

            vertice[depth]++;
            bit_test = 0;
            bit_test |= (1<<vertice[depth]);

            if(vertice[depth] == N_l){
                vertice[depth] = _EMPTY_;
                //if(block_ub > upper)   block_ub = upper;
            }else if (!(flag &  bit_test ) && GPU_queens_stillLegal(vertice, depth)){

                    ++tree_size;
                    flag |= (1ULL<<vertice[depth]);

                    depth++;

                    if (depth == N_l) { //sol
                        ++qtd_solucoes_thread; 
                    }else continue;
                }else continue;

            depth--;
            flag &= ~(1ULL<<vertice[depth]);

            }while(depth >= depthGlobal); //FIM DO DFS_BNB

        sols[idx] = qtd_solucoes_thread;
        vector_of_tree_size[idx] = tree_size;
    }//if
}//kernel
////////



unsigned long long int BP_queens_prefixes(int size, int initialDepth ,unsigned long long *tree_size, QueenRoot *root_prefixes){

    unsigned int flag = 0;
    int bit_test = 0;
    char vertice[20]; //representa o ciclo
    int i, depth; //para dizer que 0-1 ja foi visitado e a busca comeca de 1, bote 2
    unsigned long long int local_tree = 0ULL;
    unsigned long long int num_sol = 0;
   //register int custo = 0;

    /*initialization*/
    for (i = 0; i < size; ++i) { //
        vertice[i] = -1;
    }

    depth = 0;

    do{

        vertice[depth]++;
        bit_test = 0;
        bit_test |= (1<<vertice[depth]);


        if(vertice[depth] == size){
            vertice[depth] = _EMPTY_;
                //if(block_ub > upper)   block_ub = upper;
        }else if ( MCstillLegal(vertice, depth) && !(flag &  bit_test ) ){ //is legal

                flag |= (1ULL<<vertice[depth]);
                depth++;
                ++local_tree;
                if (depth == initialDepth){ //handle solution
                   prefixesHandleSol(root_prefixes,flag,vertice,initialDepth,num_sol);
                   num_sol++;
            }else continue;
        }else continue;

        depth--;
        flag &= ~(1ULL<<vertice[depth]);

    }while(depth >= 0);

    *tree_size = local_tree;

    return num_sol;
}



void GPU_call_cuda_queens(short size, int initial_depth, unsigned int n_explorers, QueenRoot *root_prefixes_h ,
	unsigned long long int *vector_of_tree_size_h, unsigned long long int *sols_h, int gpu_id){
    
    hipSetDevice(gpu_id);
   // cudaFuncSetCacheConfig(BP_queens_root_dfs,cudaFuncCachePreferL1);

    unsigned long long int *vector_of_tree_size_d;
    unsigned long long int *sols_d;
    QueenRoot *root_prefixes_d;

    int num_blocks = ceil((double)n_explorers/_QUEENS_BLOCK_SIZE_);

    hipMalloc((void**) &vector_of_tree_size_d,n_explorers*sizeof(unsigned long long int));
    hipMalloc((void**) &sols_d,n_explorers*sizeof(unsigned long long int));
    hipMalloc((void**) &root_prefixes_d,n_explorers*sizeof(QueenRoot));

    //I Think this is not possible in Chapel. It must be internal
    hipMemcpy(root_prefixes_d, root_prefixes_h, n_explorers * sizeof(QueenRoot), hipMemcpyHostToDevice);

    printf("\n### Regular BP-DFS search. ###\n");
    
    //kernel_start =  rtclock();
    
    BP_queens_root_dfs<<< num_blocks,_QUEENS_BLOCK_SIZE_>>> (size,n_explorers,initial_depth,root_prefixes_d, vector_of_tree_size_d,sols_d);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    //kernel_stop = rtclock();

    hipMemcpy(vector_of_tree_size_h,vector_of_tree_size_d,n_explorers*sizeof(unsigned long long int),hipMemcpyDeviceToHost);
    hipMemcpy(sols_h,sols_d,n_explorers*sizeof(unsigned long long int),hipMemcpyDeviceToHost);
    
    hipFree(vector_of_tree_size_d);
    hipFree(sols_d);
    hipFree(root_prefixes_d);
    //After that, Chapel reduces the values
}


int main(int argc, char *argv[]){


    int initialDepth;
    int size;
    unsigned long long initial_tree_size = 0ULL;
    unsigned long long qtd_sols_global = 0ULL;
    unsigned long long gpu_tree_size = 0ULL;

    initialDepth = atoi(argv[2]);
    size = atoi(argv[1]);
    unsigned int nMaxPrefixos = 75580635;

    printf("\n### Initial depth: %d - Size: %d:", initialDepth, size);

    QueenRoot* root_prefixes_h = (QueenRoot*)malloc(sizeof(QueenRoot)*nMaxPrefixos);
    unsigned long long int *vector_of_tree_size_h = (unsigned long long int*)malloc(sizeof(unsigned long long int)*nMaxPrefixos);
    unsigned long long int *solutions_h = (unsigned long long int*)malloc(sizeof(unsigned long long int)*nMaxPrefixos);

    //initial search, getting the tree root nodes for the gpu;
    unsigned long long n_explorers = BP_queens_prefixes((short)size, initialDepth ,&initial_tree_size, root_prefixes_h);

    //calling the gpu-based search
    GPU_call_cuda_queens((short)size, initialDepth, n_explorers, root_prefixes_h ,vector_of_tree_size_h, solutions_h, 0);

 printf("\nInitial tree size: %llu", initial_tree_size );

    for(int i = 0; i<n_explorers;++i){
        if(solutions_h[i]>0)
            qtd_sols_global += solutions_h[i];
        if(vector_of_tree_size_h[i]>0) 
            gpu_tree_size +=vector_of_tree_size_h[i];

    }

    printf("\nGPU Tree size: %llu\nTotal tree size: %llu\nNumber of solutions found: %llu.\n", gpu_tree_size,(initial_tree_size+gpu_tree_size),qtd_sols_global );
    return 0;
}  