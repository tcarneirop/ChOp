
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <sys/time.h>
#include <omp.h>

#define CUDA_QUEENS_BLOCK_SIZE_ 128
#define _EMPTY_      -1


double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}

typedef struct queen_root{
    unsigned int control;
    int8_t board[12]; //maximum depth of the solution space.
} QueenRoot;


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


void  get_load_each_gpu(unsigned long long gpu_load, int num_gpus, unsigned long long *device_load){

    for(int device = 0; device<num_gpus;++device){
        device_load[device] = gpu_load/num_gpus;
        if(device == (num_gpus-1)){
            device_load[device]+= gpu_load%num_gpus;
        }
    }
}//////


__device__  inline bool GPU_queens_stillLegal(const char *__restrict__ board, const int r){

  bool safe = true;
  int i, rev_i, offset;
  const char base = board[r];
  // Check vertical
  for ( i = 0, rev_i = r-1, offset=1; i < r; ++i, --rev_i, offset++)
    safe &= !((board[i] == base) | ( (board[rev_i] == base-offset) |
                                     (board[rev_i] == base+offset)));
  return safe;
}

inline bool MCstillLegal(const char *board, const int r)
{

    int i;
    int ld;
    int rd;
    // Check vertical
    for ( i = 0; i < r; ++i)
        if (board[i] == board[r]) return false;
    // Check diagonals
    ld = board[r];  //left diagonal columns
    rd = board[r];  // right diagonal columns
    for ( i = r-1; i >= 0; --i) {
      --ld; ++rd;
      if (board[i] == ld || board[i] == rd) return false;
    }

    return true;
}


inline void prefixesHandleSol(QueenRoot *root_prefixes,unsigned int flag,char *board,int initialDepth,int num_sol){

    root_prefixes[num_sol].control = flag;

    for(int i = 0; i<initialDepth;++i)
      root_prefixes[num_sol].board[i] = (char)board[i];
}


unsigned long long int BP_queens_prefixes(int size, int initialDepth,
    unsigned long long *tree_size, QueenRoot *root_prefixes){

    unsigned int flag = 0;
    int bit_test = 0;
    char board[32]; 
    int i, depth; 
    unsigned long long int local_tree = 0ULL;
    unsigned long long int num_sol = 0;

    #ifdef IMPROVED
    uint break_cond =  (size/2) + (size & 1);
    #endif 

    /*initialization*/
    for (i = 0; i < size; ++i) { //
        board[i] = -1;
    }

    depth = 0;

    do{

        board[depth]++;
        bit_test = 0;
        bit_test |= (1<<board[depth]);


        if(board[depth] == size){
            board[depth] = _EMPTY_;
                //if(block_ub > upper)   block_ub = upper;
        }else if ( MCstillLegal(board, depth) && !(flag &  bit_test ) ){ //it is a valid subsol 
   
           #ifdef IMPROVED
            if(depth == 1){

                if(size& 1){
                    if (board[0] == break_cond-1 && board[1] > board[0]) 
                        break;
                }
                else{
                    if (board[0] == break_cond)
                        break;
                }
            }
            #endif 

                flag |= (1ULL<<board[depth]);
                depth++;
                ++local_tree;
                if (depth == initialDepth){ //handle solution
                   prefixesHandleSol(root_prefixes,flag,board,initialDepth,num_sol);
                   num_sol++;
            }else continue;
        }else continue;

        depth--;
        flag &= ~(1ULL<<board[depth]);

    }while(depth >= 0);

    *tree_size = local_tree;

    return num_sol;
}

__global__ void BP_queens_root_dfs( const int N, const unsigned int nPrefixes, 
    const int initial_depth,
    QueenRoot *__restrict__ root_prefixes,
    unsigned long long int *__restrict__ vector_of_tree_size, 
    unsigned long long int *__restrict__ sols){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
       
    if (idx < nPrefixes) {
        unsigned int flag = 0;
        char board[32];
        int N_l = N;
        int i, depth;
        unsigned long long  qtd_sols_thread = 0ULL;
        int depthGlobal = initial_depth;
        unsigned long long int tree_size = 0ULL;

        for (i = 0; i < N_l; ++i) {
            board[i] = _EMPTY_;
        }

        flag = root_prefixes[idx].control;

        for (i = 0; i < depthGlobal; ++i)
            board[i] = root_prefixes[idx].board[i];

        depth=depthGlobal;

        do{

            board[depth]++;
            const int mask = 1<<board[depth];

            if(board[depth] == N_l){
                board[depth] = _EMPTY_;
                depth--;
                flag &= ~(1<<board[depth]);
            }else if (!(flag &  mask ) && GPU_queens_stillLegal(board, depth)){

                    ++tree_size;
                    flag |= mask;

                    depth++;

                    if (depth == N_l) { //sol
                        ++qtd_sols_thread ;

                        depth--;
                        flag &= ~mask;
                    }
                }
            }while(depth >= depthGlobal); //FIM DO DFS_BNB

        sols[idx] = qtd_sols_thread ;
        vector_of_tree_size[idx] = tree_size;
    }//if
}//kernel
////////



void CUDA_call_queens(int size, int initial_depth, unsigned int n_explorers, QueenRoot *root_prefixes_h ,
	unsigned long long *vector_of_tree_size_h, unsigned long long *sols_h, int gpu_id, int block_size){
     
    hipSetDevice(gpu_id);

    unsigned long long *vector_of_tree_size_d;
    unsigned long long *sols_d;
    QueenRoot *root_prefixes_d;

    int num_blocks = ceil((double)n_explorers/block_size);

    hipMalloc((void**) &vector_of_tree_size_d,n_explorers*sizeof(unsigned long long));
    hipMalloc((void**) &sols_d,n_explorers*sizeof(unsigned long long));
    hipMalloc((void**) &root_prefixes_d,n_explorers*sizeof(QueenRoot));

    hipMemcpy(root_prefixes_d, root_prefixes_h, n_explorers * sizeof(QueenRoot), hipMemcpyHostToDevice);
    
    BP_queens_root_dfs<<< num_blocks,block_size>>>(size,n_explorers,initial_depth,root_prefixes_d, vector_of_tree_size_d,sols_d);
   
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    hipMemcpy(vector_of_tree_size_h,vector_of_tree_size_d,n_explorers*sizeof(unsigned long long),hipMemcpyDeviceToHost);
    hipMemcpy(sols_h,sols_d,n_explorers*sizeof(unsigned long long),hipMemcpyDeviceToHost);
    
}

void call_queens(int size, int initialDepth, int block_size){


    unsigned long long initial_tree_size = 0ULL;
    unsigned long long qtd_sols_global = 0ULL;
    unsigned long long gpu_tree_size = 0ULL;

    unsigned int nMaxPrefixes = 75580635;
    int num_gpus = 0;
    hipGetDeviceCount( &num_gpus );
    printf("\nNumber of GPUS: %d\n", num_gpus );


    unsigned long long device_load[num_gpus];
  
   
    QueenRoot* root_prefixes_h = (QueenRoot*)malloc(sizeof(QueenRoot)*nMaxPrefixes);
    unsigned long long int *vector_of_tree_size_h = (unsigned long long int*)malloc(sizeof(unsigned long long int)*nMaxPrefixes);
    unsigned long long int *solutions_h = (unsigned long long int*)malloc(sizeof(unsigned long long int)*nMaxPrefixes);


    double initial_time = rtclock();

    //initial search, getting Feasible, Valid and Incomplete solutions -- subproblems;
    unsigned long long n_explorers = BP_queens_prefixes((short)size, initialDepth ,&initial_tree_size, root_prefixes_h);

    printf("\n### Queens size: %d, Initial depth: %d, Block size: %d - Num_explorers: %llu", size, initialDepth, block_size,n_explorers);

    get_load_each_gpu(n_explorers, num_gpus, device_load);
    printf("\nLoad of each GPU:");
    for(int device = 0; device<num_gpus;++device){
        printf("\n\tDevice: %d - load : %llu ", device, device_load[device]);
    }
    printf("\n\n");

    //calling the gpu-based search
    omp_set_num_threads(num_gpus);

    #pragma omp parallel for default(none) shared(size, num_gpus, n_explorers, block_size, initialDepth, device_load, root_prefixes_h, vector_of_tree_size_h, solutions_h)
    for(uint device = 0; device<num_gpus; ++device){
  
        unsigned long long local_stride =  device * (n_explorers/num_gpus);
        printf("\n\tNum threads: %d, Thread: %d - Device: %d - load : %llu ", omp_get_num_threads(),omp_get_thread_num(),  device, device_load[device]);
        CUDA_call_queens(size, initialDepth,device_load[device], root_prefixes_h+local_stride,vector_of_tree_size_h+local_stride, solutions_h+local_stride, device, block_size);
    
    } 
   
    double final_time = rtclock();

    //Reducing the metrics
    for(int i = 0; i<n_explorers;++i){
        qtd_sols_global += solutions_h[i];
        gpu_tree_size +=vector_of_tree_size_h[i];
    }

    #ifdef IMPROVED
       qtd_sols_global*=2;
    #endif


    printf("\nInitial tree size: %llu", initial_tree_size );
    printf("\nGPU Tree size: %llu\nTotal tree size: %llu\nNumber of solutions found: %llu.\n", gpu_tree_size,(initial_tree_size+gpu_tree_size),qtd_sols_global );
    printf("\nElapsed total: %.3f\n", (final_time-initial_time));

}


int main(int argc, char *argv[]){

    hipFree(0);
    int block_size;
    int initialDepth;
    int size;

    #ifdef IMPROVED
        printf("### IMPROVED SEARCH - Avoiding mirrored solutions\n");
    #endif


    size = atoi(argv[1]);
    initialDepth = atoi(argv[2]);
    block_size   =   atoi(argv[3]);

    call_queens(size, initialDepth, block_size);

    return 0;
}
